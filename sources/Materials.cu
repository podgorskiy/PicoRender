#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_cuda.h>
#include "ray_phisics.h"
#include "ray_payload.h"
#include "sampling.h"

/*! the implicit state's ray we will intersect against */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
/*! the per ray data we operate on */
rtDeclareVariable(RayPayload, ray_payload, rtPayload, );
rtDeclareVariable(rtObject, world, , );

/*! the attributes we use to communicate between intersection programs and hit program */
rtDeclareVariable(float3, hit_rec_normal, attribute hit_rec_normal, );
rtDeclareVariable(float3, hit_rec_p, attribute hit_rec_p, );


/*! and finally - that particular material's parameters */
rtDeclareVariable(float3, albedo, , );


RT_PROGRAM void lambertian_hit()
{
    ray_payload.scatterEvent = RayPayload::rayGotBounced;
    ray_payload.direction = lambert_no_tangent(to_glm(hit_rec_normal), ray_payload.rs);
    ray_payload.origin = to_glm(hit_rec_p);
    ray_payload.attenuation = to_glm(albedo);
}
