#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_cuda.h>
#include "ray_phisics.h"
#include "ray_payload.h"
#include "sampling.h"

/*! the implicit state's ray we will intersect against */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
/*! the per ray data we operate on */
rtDeclareVariable(RayPayload, ray_payload, rtPayload, );
rtDeclareVariable(rtObject, world, , );

/*! the attributes we use to communicate between intersection programs and hit program */
rtDeclareVariable(float3, hit_rec_normal, attribute hit_rec_normal, );
rtDeclareVariable(float3, hit_rec_p, attribute hit_rec_p, );

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, hit_point, attribute hit_point, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );

/*! and finally - that particular material's parameters */
rtDeclareVariable(float3, albedo, , );


RT_PROGRAM void lambertian_hit()
{
    const vec3 normal = faceforward( shading_normal, -ray.direction, geometric_normal );

    ray_payload.scatterEvent = RayPayload::rayGotBounced;
    ray_payload.direction = lambert_no_tangent(normal, ray_payload.rs);
    ray_payload.origin = hit_point;
    ray_payload.attenuation = albedo;
}
