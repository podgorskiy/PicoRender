#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_cuda.h>
#include "ray_phisics.h"
#include "ray_payload.h"
#include "sampling.h"

/*! the implicit state's ray we will intersect against */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
/*! the per ray data we operate on */
rtDeclareVariable(RayPayload, ray_payload, rtPayload, );
rtDeclareVariable(rtObject, world, , );

/*! the attributes we use to communicate between intersection programs and hit program */
rtDeclareVariable(float3, hit_rec_normal, attribute hit_rec_normal, );
rtDeclareVariable(float3, hit_rec_p, attribute hit_rec_p, );

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, hit_point, attribute hit_point, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(int,    uv_pass,          attribute uv_pass, );
rtTextureSampler<float4, 2> albedo_texture;

/*! and finally - that particular material's parameters */
rtDeclareVariable(float3, albedo, , );


RT_PROGRAM void lambertian_hit()
{
    vec3 normal;
    if (uv_pass == 0)
    {
        normal = faceforward(shading_normal, -ray.direction, geometric_normal);
    }
    else
    {
        normal = shading_normal;
    }
    ray_payload.scatterEvent = RayPayload::rayGotBounced;
    ray_payload.direction = lambert_no_tangent(normal, ray_payload.rs);
    ray_payload.origin = hit_point;
    ray_payload.normal = normal;


    const vec3 Kd = make_float3( tex2D( albedo_texture, texcoord.x, texcoord.y ) );

    ray_payload.attenuation = Kd;
}
