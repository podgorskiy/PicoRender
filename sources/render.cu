#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "ray.h"
#include "sampling.h"

/*! the 'builtin' launch index we need to render a frame */
rtDeclareVariable(uint2, pixelID,   rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim,   );

/*! the ray related state */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData, prd, rtPayload, );

/*! the 2D, float3-type color frame buffer we'll write into */
rtBuffer<float3, 2> fb;

rtDeclareVariable(int, numSamples, , );

rtDeclareVariable(rtObject, world, , );

rtDeclareVariable(float3, camera_lower_left_corner, , );
rtDeclareVariable(float3, camera_horizontal, , );
rtDeclareVariable(float3, camera_vertical, , );
rtDeclareVariable(float3, camera_origin, , );
rtDeclareVariable(float3, camera_u, , );
rtDeclareVariable(float3, camera_v, , );
rtDeclareVariable(float, camera_lens_radius, , );


/*! the actual ray generation program - note this has no formal
  function parameters, but gets its paramters throught the 'pixelID'
  and 'pixelBuffer' variables/buffers declared above */
RT_PROGRAM void Render()
{
    uint32_t pixel_index = pixelID.y * launchDim.x + pixelID.x;
    vec3 col(0.f, 0.f, 0.f);
    rnd::RandomState rs(pixel_index);

    for (int s = 0; s < numSamples; s++)
    {
        float u = float(pixelID.x + rs.rand1()) / float(launchDim.x);
        float v = float(pixelID.y + rs.rand1()) / float(launchDim.y);
        // optix::Ray ray = Camera::generateRay(u, v, rnd);
        // col += color(ray, rnd);
        col += vec3(u, v, 0.);
    }
    col = col / scalar(numSamples);

    fb[pixelID] = to_cuda(col);
}
