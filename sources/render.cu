#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "ray_payload.h"
#include "sampling.h"
#include "Camera.h"

/*! the 'builtin' launch index we need to render a frame */
rtDeclareVariable(uint2, pixelID,   rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim,   );

/*! the ray related state */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayPayload, ray_payload, rtPayload, );

rtDeclareVariable(rtObject, root, , );

rtBuffer<float4, 2> pixelBuffer;

rtDeclareVariable(int,      numSamples, , );
rtDeclareVariable(int,      bounces, , );
rtDeclareVariable(float3,   camera_origin, , );
rtDeclareVariable(float3,   camera_lookat, , );
rtDeclareVariable(float3,   camera_up, , );
rtDeclareVariable(float,    camera_vfov, , );
rtDeclareVariable(float,    camera_aperture, , );
rtDeclareVariable(float,    camera_focusDist, , );



inline __device__ vec3 missColor(const optix::Ray &ray)
{
  const vec3 unit_direction = normalize(to_glm(ray.direction));
  const scal t = 0.5*(unit_direction.y + 1.0);
  const vec3 c = (scal(1.0) - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
  return c;
}


inline __device__ vec4 ComputeBounces(optix::Ray &ray, rnd::RandomState &rs)
{
    RayPayload ray_payload;

	vec3 light(0.0);
	vec3 color(1.0);
    int k = 0;
    for (; k < bounces; ++k)
    {
        ray_payload.rs = &rs;
        rtTrace(root, ray, ray_payload);
        if (ray_payload.scatterEvent == RayPayload::rayDidntHitAnything)
        {
            light = missColor(ray);
            break;
        }
        else if (length(color) < 0.01 || ray_payload.scatterEvent == RayPayload::rayGotCancelled)
        {
            break;
        }
        else
        {
            color *= ray_payload.attenuation;
            ray = optix::make_Ray(
                    /* origin   : */ to_cuda(ray_payload.origin),
                    /* direction: */ to_cuda(ray_payload.direction),
                    /* ray type : */ 0,
                    /* tmin     : */ 1e-3f,
                    /* tmax     : */ RT_DEFAULT_MAX);
        }
    }
    if (k != 0)
    {
        return vec4(light * color, 1.);
    }
    else
    {
        return vec4(0.);
    }
}


RT_PROGRAM void Render()
{
    uint32_t pixel_index = pixelID.y * launchDim.x + pixelID.x;

    vec4 col(0.f, 0.f, 0.f, 0.f);
    rnd::RandomState rs(pixel_index);

    float aspect = float(launchDim.x) / float(launchDim.y);

    Camera camera = Camera(
            to_glm(camera_origin),
            to_glm(camera_lookat),
            to_glm(camera_up),
            camera_vfov, aspect, camera_aperture, camera_focusDist);

    for (int s = 0; s < numSamples; s++)
    {
        int y_id = launchDim.y - pixelID.y - 1;
        float u = float(pixelID.x + rs.rand1()) / float(launchDim.x);
        float v = float(y_id + rs.rand1()) / float(launchDim.y);

        optix::Ray ray = camera.generateRay(vec2(u, v), rs);

        col += ComputeBounces(ray, rs);
        // col += missColor(ray);
    }
    col = col / scal(numSamples);

    pixelBuffer[pixelID] = to_cuda(vec4(pow(vec3(col), vec3(1.0 / 2.2)), col.a));
}

RT_PROGRAM void Miss()
{
    ray_payload.scatterEvent = RayPayload::rayDidntHitAnything;
}
