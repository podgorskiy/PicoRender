#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "ray_payload.h"
#include "sampling.h"
#include "Camera.h"

/*! the 'builtin' launch index we need to render a frame */
rtDeclareVariable(uint2, pixelID,   rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim,   );

/*! the ray related state */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayPayload, ray_payload, rtPayload, );

rtDeclareVariable(rtObject, world, , );


rtBuffer<float3, 2> fb;
rtDeclareVariable(int, numSamples, , );
rtDeclareVariable(float3, camera_origin, , );
rtDeclareVariable(float3, camera_lookat, , );
rtDeclareVariable(float3, camera_up, , );
rtDeclareVariable(float,  camera_vfov, , );
rtDeclareVariable(float,  camera_aperture, , );
rtDeclareVariable(float,  camera_focusDist, , );



inline __device__ vec3 missColor(const optix::Ray &ray)
{
  const vec3 unit_direction = normalize(to_glm(ray.direction));
  const scal t = 0.5*(unit_direction.y + 1.0);
  const vec3 c = (scal(1.0) - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
  return c;
}


inline __device__ vec3 ComputeBounces(optix::Ray &ray, rnd::RandomState &rs)
{
    RayPayload ray_payload;
    vec3 attenuation = vec3(1.);

    for (int k = 0; k < 5; ++k)
    {
        ray_payload.rs = &rs;
        rtTrace(world, ray, ray_payload);
        if (ray_payload.scatterEvent == RayPayload::rayDidntHitAnything)
        {
            return attenuation * missColor(ray);
        }
        else if (length(attenuation) < 0.01 || ray_payload.scatterEvent == RayPayload::rayGotCancelled)
        {
            return vec3(0.);
        }
        else { // ray is still alive, and got properly bounced
            attenuation *= ray_payload.attenuation;
            ray = optix::make_Ray(
                    /* origin   : */ to_cuda(ray_payload.origin),
                    /* direction: */ to_cuda(ray_payload.direction),
                    /* ray type : */ 0,
                    /* tmin     : */ 1e-3f,
                    /* tmax     : */ RT_DEFAULT_MAX);
        }
    }
    // recursion did not terminate - cancel it
    return vec3(0.);
}


RT_PROGRAM void Render()
{
    uint32_t pixel_index = pixelID.y * launchDim.x + pixelID.x;

    vec3 col(0.f, 0.f, 0.f);
    rnd::RandomState rs(pixel_index);

    float aspect = float(launchDim.x) / float(launchDim.y);

    Camera camera = Camera(
            to_glm(camera_origin),
            to_glm(camera_lookat),
            to_glm(camera_up),
            camera_vfov, aspect, camera_aperture, camera_focusDist);

    for (int s = 0; s < numSamples; s++)
    {
        int y_id = launchDim.y - pixelID.y - 1;
        float u = float(pixelID.x + rs.rand1()) / float(launchDim.x);
        float v = float(y_id + rs.rand1()) / float(launchDim.y);

        optix::Ray ray = camera.generateRay(vec2(u, v), rs);

        col += ComputeBounces(ray, rs);
        // col += missColor(ray);
    }
    col = col / scal(numSamples);

    fb[pixelID] = to_cuda(pow(col, vec3(1.0 / 2.2)));
}

RT_PROGRAM void Miss()
{
    ray_payload.scatterEvent = RayPayload::rayDidntHitAnything;
}
