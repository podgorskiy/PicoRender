#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "ray_payload.h"
#include "sampling.h"
#include "Camera.h"

/*! the 'builtin' launch index we need to render a frame */
rtDeclareVariable(uint2, pixelID,   rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim,   );

/*! the ray related state */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayPayload, ray_payload, rtPayload, );

rtDeclareVariable(rtObject, root, , );

rtBuffer<float4, 2> pixelBuffer;

rtDeclareVariable(int,      numSamples, , );
rtDeclareVariable(int,      bounces, , );
rtDeclareVariable(float3,   camera_origin, , );
rtDeclareVariable(float3,   camera_lookat, , );
rtDeclareVariable(float3,   camera_up, , );
rtDeclareVariable(float,    camera_vfov, , );
rtDeclareVariable(float,    camera_aperture, , );
rtDeclareVariable(float,    camera_focusDist, , );



inline __device__ vec3 missColor(const optix::Ray &ray)
{
    const vec3 unit_direction = normalize(ray.direction);
    const scal t = 0.5 * (unit_direction.y + 1.0);
    const vec3 c = (scal(1.0) - t) * make_float3(1.0, 1.0, 1.0) + t * make_float3(0.5, 0.7, 1.0);
    return c;
}


inline __device__ vec4 ComputeBounces(optix::Ray &ray, rnd::RandomState &rs)
{
    RayPayload ray_payload;

	vec3 light = make_float3(0.0);
	vec3 color = make_float3(1.0);
    int k = 0;
    for (; k < bounces; ++k)
    {
        ray_payload.rs = &rs;
        rtTrace(root, ray, ray_payload);
        if (ray_payload.scatterEvent == RayPayload::rayDidntHitAnything)
        {
            light = missColor(ray);
            break;
        }
        else if (length(color) < 0.01 || ray_payload.scatterEvent == RayPayload::rayGotCancelled)
        {
            break;
        }
        else
        {
            color *= ray_payload.attenuation;
            ray = optix::make_Ray(
                    /* origin   : */ ray_payload.origin,
                    /* direction: */ ray_payload.direction,
                    /* ray type : */ 0,
                    /* tmin     : */ 1e-3f,
                    /* tmax     : */ RT_DEFAULT_MAX);
        }
    }
    if (k != 0)
    {
        return make_float4(light * color, 1.);
    }
    else
    {
        return make_float4(0.);
    }
}


RT_PROGRAM void Render()
{
    uint32_t pixel_index = pixelID.y * launchDim.x + pixelID.x;

    vec4 col = make_float4(0.f, 0.f, 0.f, 0.f);
    rnd::RandomState rs(pixel_index, pixel_index*pixel_index);

    float aspect = float(launchDim.x) / float(launchDim.y);

    Camera camera = Camera(
            camera_origin,
            camera_lookat,
            camera_up,
            camera_vfov, aspect, camera_aperture, camera_focusDist);

    for (int s = 0; s < numSamples; s++)
    {
        int y_id = launchDim.y - pixelID.y - 1;
        float u = float(pixelID.x + rs.rand1()) / float(launchDim.x);
        float v = float(y_id + rs.rand1()) / float(launchDim.y);

        optix::Ray ray = camera.generateRay(make_float2(u, v), rs);

        col += ComputeBounces(ray, rs);
        // col += missColor(ray);
    }
    col = col / scal(numSamples);
    col.x = powf(col.x, 1.0/ 2.2);
    col.y = powf(col.y, 1.0/ 2.2);
    col.z = powf(col.z, 1.0/ 2.2);
    pixelBuffer[pixelID] = col;
}

RT_PROGRAM void Miss()
{
    ray_payload.scatterEvent = RayPayload::rayDidntHitAnything;
}
