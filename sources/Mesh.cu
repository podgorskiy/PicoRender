#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtBuffer<float3> vertex_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<int3>   index_buffer;
rtBuffer<int>    material_buffer;

rtDeclareVariable(float3, texcoord,         attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, );

rtDeclareVariable(float3, hit_point,        attribute hit_point, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void mesh_intersect(int primIdx) {
    const int3 v_0 = index_buffer[3 * primIdx + 0];
    const int3 v_1 = index_buffer[3 * primIdx + 1];
    const int3 v_2 = index_buffer[3 * primIdx + 2];

    const float3 p0 = vertex_buffer[v_0.x];
    const float3 p1 = vertex_buffer[v_1.x];
    const float3 p2 = vertex_buffer[v_2.x];

    // Intersect ray with triangle
    float3 n;
    float t, beta, gamma;
    if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma)) {

        if (rtPotentialIntersection(t)) {

            geometric_normal = normalize(n);
            if (normal_buffer.size() == 0) {
                shading_normal = geometric_normal;
            } else {
                float3 n0 = normal_buffer[v_0.y];
                float3 n1 = normal_buffer[v_1.y];
                float3 n2 = normal_buffer[v_2.y];
                shading_normal = normalize(n1 * beta + n2 * gamma + n0 * (1.0f - beta - gamma));
            }

            if (texcoord_buffer.size() == 0)
            {
                texcoord = make_float3(0.0f, 0.0f, 0.0f);
            } else {
                float2 t0 = texcoord_buffer[v_0.z];
                float2 t1 = texcoord_buffer[v_1.z];
                float2 t2 = texcoord_buffer[v_2.z];
                texcoord = make_float3(t1 * beta + t2 * gamma + t0 * (1.0f - beta - gamma));
            }
            float3 original_hit_point = ray.origin + t * ray.direction;

            float  refined_t = -(optix::dot(geometric_normal, original_hit_point - p0)) / optix::dot(geometric_normal, ray.direction);
            float3 refined_hit_point = original_hit_point + refined_t * ray.direction;

            hit_point = refined_hit_point;

            rtReportIntersection(material_buffer[primIdx]);
        }
    }
}


RT_PROGRAM void mesh_bounds(int primIdx, float result[6]) {
    const int3 v_0 = index_buffer[3 * primIdx + 0];
    const int3 v_1 = index_buffer[3 * primIdx + 1];
    const int3 v_2 = index_buffer[3 * primIdx + 2];

    const float3 v0 = vertex_buffer[v_0.x];
    const float3 v1 = vertex_buffer[v_1.x];
    const float3 v2 = vertex_buffer[v_2.x];
    const float area = length(cross(v1 - v0, v2 - v0));

    optix::Aabb *aabb = (optix::Aabb *) result;

    if (area > 0.0f && !isinf(area)) {
        aabb->m_min = fminf(fminf(v0, v1), v2);
        aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
    } else {
        aabb->invalidate();
    }
}
