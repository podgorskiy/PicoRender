#include "hip/hip_runtime.h"
#include "types.h"
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

rtBuffer<float3> vertex_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<int3>   index_buffer;
rtBuffer<int>    material_buffer;

rtDeclareVariable(float3, texcoord,         attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, );

rtDeclareVariable(float3, hit_point,        attribute hit_point, );
rtDeclareVariable(int,    uv_pass,          attribute uv_pass, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void mesh_intersect_uv(int primIdx) {
    const int3 v_0 = index_buffer[3 * primIdx + 0];
    const int3 v_1 = index_buffer[3 * primIdx + 1];
    const int3 v_2 = index_buffer[3 * primIdx + 2];

    const vec3 _p0 = vertex_buffer[v_0.x];
    const vec3 _p1 = vertex_buffer[v_1.x];
    const vec3 _p2 = vertex_buffer[v_2.x];
    const vec3 p0 = make_float3(texcoord_buffer[v_0.z], 0.);
    const vec3 p1 = make_float3(texcoord_buffer[v_1.z], 0.);
    const vec3 p2 = make_float3(texcoord_buffer[v_2.z], 0.);

    // Intersect ray with triangle
    vec3 n;
    float t, beta, gamma;
    if (intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma)) {

        if (rtPotentialIntersection(t)) {
            geometric_normal = normalize(cross(_p1 - _p0, _p2 - _p0));
            if (normal_buffer.size() == 0) {
                shading_normal = geometric_normal;
            } else {
                vec3 n0 = normal_buffer[v_0.y];
                vec3 n1 = normal_buffer[v_1.y];
                vec3 n2 = normal_buffer[v_2.y];
                shading_normal = normalize(n1 * beta + n2 * gamma + n0 * (1.0f - beta - gamma));
            }
            hit_point = _p1 * beta + _p2 * gamma + _p0 * (1.0f - beta - gamma);

            if (texcoord_buffer.size() == 0)
            {
                texcoord = make_float3(0.0f, 0.0f, 0.0f);
            } else {
                vec2 t0 = texcoord_buffer[v_0.z];
                vec2 t1 = texcoord_buffer[v_1.z];
                vec2 t2 = texcoord_buffer[v_2.z];
                texcoord = make_float3(t1 * beta + t2 * gamma + t0 * (1.0f - beta - gamma));
            }
            uv_pass = 1;

            rtReportIntersection(material_buffer[primIdx]);
        }
    }
}


RT_PROGRAM void mesh_bounds_uv(int primIdx, float result[6]) {
    const int3 v_0 = index_buffer[3 * primIdx + 0];
    const int3 v_1 = index_buffer[3 * primIdx + 1];
    const int3 v_2 = index_buffer[3 * primIdx + 2];

    vec3 v0 = make_float3(texcoord_buffer[v_0.z], 0.);
    vec3 v1 = make_float3(texcoord_buffer[v_1.z], 0.);
    vec3 v2 = make_float3(texcoord_buffer[v_2.z], 0.);

    const float area = length(cross(v1 - v0, v2 - v0));

    optix::Aabb *aabb = (optix::Aabb *) result;

    if (area > 0.0f && !isinf(area)) {
        aabb->m_min = fminf(fminf(v0, v1), v2);
        aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
    } else {
        aabb->invalidate();
    }
}
