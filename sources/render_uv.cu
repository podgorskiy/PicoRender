#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "ray_payload.h"
#include "sampling.h"
#include "Camera.h"

/*! the 'builtin' launch index we need to render a frame */
rtDeclareVariable(uint2, pixelID,   rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim,   );

/*! the ray related state */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayPayload, ray_payload, rtPayload, );

rtDeclareVariable(rtObject, root, , );
rtDeclareVariable(rtObject, root_uv, , );

rtBuffer<float4, 2> albedoBuffer;
rtBuffer<float4, 2> normalBuffer;
rtBuffer<float4, 2> bentNormalBuffer;
rtBuffer<float4, 2> giBuffer;
rtBuffer<float4, 2> finalBuffer;

rtDeclareVariable(int,      numSamples, , );
rtDeclareVariable(int,      bounces, , );
rtDeclareVariable(float3,   camera_origin, , );
rtDeclareVariable(float3,   camera_lookat, , );
rtDeclareVariable(float3,   camera_up, , );
rtDeclareVariable(float,    camera_vfov, , );
rtDeclareVariable(float,    camera_aperture, , );
rtDeclareVariable(float,    camera_focusDist, , );



inline __device__ vec3 missColor(const optix::Ray &ray)
{
    return make_float3(1.0);
}


inline __device__ vec4 Radiance(optix::Ray &ray, rnd::RandomState &rs, vec3& albedo_out, vec3& normal_out, vec3& bent_normal_out)
{
    RayPayload ray_payload;

	vec3 light = make_float3(0.0);
	vec3 color = make_float3(1.0);
	vec3 attenuation;

    int k = 0;
    for (; k < bounces; ++k)
    {
        ray_payload.rs = &rs;
        rtTrace(k == 0 ? root_uv : root, ray, ray_payload);
        if (ray_payload.scatterEvent == RayPayload::rayDidntHitAnything)
        {
            light = missColor(ray);
            break;
        }
        else if (ray_payload.scatterEvent == RayPayload::rayGotBounced)
        {
			attenuation = ray_payload.attenuation;
            //color *= ray_payload.attenuation;
            ray = optix::make_Ray(
                    /* origin   : */ ray_payload.origin,
                    /* direction: */ ray_payload.direction,
                    /* ray type : */ 0,
                    /* tmin     : */ 1e-3f,
                    /* tmax     : */ RT_DEFAULT_MAX);

			if (k == 0)
			{
				albedo_out = attenuation;
				normal_out = ray_payload.normal;
				attenuation = make_float3(1.0);
				bent_normal_out = ray_payload.direction;
			}
            color = color * attenuation;

            if (length(color) < 0.01)
            {
                break;
            }
        }
        else
        {
            color = make_float3(0.0);
            break;
        }
    }
    if (k != 0)
    {
        vec3 lightColor = light * color;
		bent_normal_out *= dot(lightColor, make_float3(0.3, 0.59, 0.11));
        return make_float4(lightColor, 1.);
    }
    else
    {
		albedo_out = make_float3(0.);
		normal_out = make_float3(0.);
		bent_normal_out = make_float3(0.);
        return make_float4(0.);
    }
}


__device__ vec3 pow(vec3 x, float e)
{
    return make_float3(powf(x.x, e), powf(x.y, e), powf(x.z, e));
}

__device__ vec4 pow(vec4 x, float e)
{
    return make_float4(powf(x.x, e), powf(x.y, e), powf(x.z, e), x.w);
}


RT_PROGRAM void RenderUV()
{
    uint32_t pixel_index = pixelID.y * launchDim.x + pixelID.x;

    vec4 col = make_float4(0.f, 0.f, 0.f, 0.f);
    rnd::RandomState rs(pixel_index, pixel_index*pixel_index);

    float aspect = float(launchDim.x) / float(launchDim.y);

    Camera camera = Camera(
            camera_origin,
            camera_lookat,
            camera_up,
            camera_vfov, aspect, camera_aperture, camera_focusDist);

    vec3 albedo = make_float3(0.);
    vec3 normal = make_float3(0.);
    vec3 bent_normal = make_float3(0.);

    for (int s = 0; s < numSamples; s++)
    {
        int y_id = launchDim.y - pixelID.y - 1;
        float u = float(pixelID.x + rs.rand1()) / float(launchDim.x);
        float v = float(y_id + rs.rand1()) / float(launchDim.y);

        optix::Ray ray = optix::make_Ray(
                /* origin   : */ make_float3(u, v, -1.0),
                /* direction: */ make_float3(0, 0, 1.),
                /* ray type : */ 0,
                /* tmin     : */ 1e-6f,
                /* tmax     : */ RT_DEFAULT_MAX);

        vec3 albedo_out;
        vec3 normal_out;
        vec3 bent_normal_out;

        col += Radiance(ray, rs, albedo_out, normal_out, bent_normal_out);
        albedo += albedo_out;
        normal += normal_out;
        bent_normal += bent_normal_out;
    }
    float c = col.w;

    vec4 final;
    if (col.w > 0.)
    {
        col /= c;
        albedo /= c;
        final = make_float4(make_float3(col.x, col.y, col.z) * albedo, 1.0);

        col = pow(col, 1.0/ 2.2);
        final = pow(final, 1.0/ 2.2);
        albedo = pow(albedo, 1.0/ 2.2);
        normal = normalize(normal);
        bent_normal = normalize(bent_normal);
    }
    else
    {
        col = make_float4(0.0);
        final = make_float4(0.0);
        albedo = make_float3(0.0);
        normal = make_float3(0.0);
        bent_normal = make_float3(0.0);
    }

    // pixelBuffer[pixelID] = col;
    albedoBuffer[pixelID] = make_float4(albedo, col.w);
    normalBuffer[pixelID] = make_float4(normal * 0.5 + make_float3(0.5), col.w);
    bentNormalBuffer[pixelID] = make_float4(bent_normal * 0.5 + make_float3(0.5), dot(make_float3(col), make_float3(0.3, 0.59, 0.11)));
    giBuffer[pixelID] = col;
    finalBuffer[pixelID] = final;
}

RT_PROGRAM void Miss()
{
    ray_payload.scatterEvent = RayPayload::rayDidntHitAnything;
}
